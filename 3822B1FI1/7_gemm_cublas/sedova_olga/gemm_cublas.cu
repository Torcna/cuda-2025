#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>

#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__    \
                      << ", code: " << err << ", reason: "                  \
                      << hipGetErrorString(err) << std::endl;              \
            throw std::runtime_error("CUDA call failed");                   \
        }                                                                   \
    } while (0)

#define CUBLAS_CHECK(call)                                                  \
    do {                                                                    \
        hipblasStatus_t status = call;                                       \
        if (status != HIPBLAS_STATUS_SUCCESS) {                             \
            std::cerr << "cuBLAS error in " << __FILE__ << ":" << __LINE__  \
                      << ", code: " << status << std::endl;                 \
            throw std::runtime_error("cuBLAS call failed");                 \
        }                                                                   \
    } while (0)

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    if (n == 0) return {};

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    size_t size = n * n * sizeof(float);
    std::vector<float> c(n * n);

    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr, *d_ct = nullptr;

    CUDA_CHECK(hipMalloc(&d_a, size));
    CUDA_CHECK(hipMalloc(&d_b, size));
    CUDA_CHECK(hipMalloc(&d_c, size));
    CUDA_CHECK(hipMalloc(&d_ct, size));

    CUBLAS_CHECK(hipblasSetMatrix(n, n, sizeof(float), a.data(), n, d_a, n));
    CUBLAS_CHECK(hipblasSetMatrix(n, n, sizeof(float), b.data(), n, d_b, n));

    const float alpha = 1.0f;
    const float beta = 0.0f;

    CUBLAS_CHECK(hipblasSgemm(handle,
                             HIPBLAS_OP_T, HIPBLAS_OP_T,
                             n, n, n,
                             &alpha,
                             d_a, n,
                             d_b, n,
                             &beta,
                             d_c, n));

     CUBLAS_CHECK(hipblasSgemm(handle,
                             HIPBLAS_OP_T, HIPBLAS_OP_N,
                             n, n, &alpha,
                             d_c, n,
                             &beta,
                             nullptr, n,
                             d_ct, n));


    CUBLAS_CHECK(hipblasGetMatrix(n, n, sizeof(float), d_ct, n, c.data(), n));

    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    CUDA_CHECK(hipFree(d_ct));
    CUBLAS_CHECK(hipblasDestroy(handle));
    return c;
}
