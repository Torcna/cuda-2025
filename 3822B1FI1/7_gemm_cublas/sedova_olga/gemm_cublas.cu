#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    if (n == 0) return {};

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    size_t size = n * n * sizeof(float);
    std::vector<float> c(n * n);

    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr, *d_ct = nullptr;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    hipMalloc(&d_ct, size);

    hipblasSetMatrix(n, n, sizeof(float), a.data(), n, d_a, n);
    hipblasSetMatrix(n, n, sizeof(float), b.data(), n, d_b, n);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_T, 
                HIPBLAS_OP_T,
                n, n, n,
                &alpha,
                d_a, n,
                d_b, n,
                &beta,
                d_c, n);

    hipblasSgemm(handle, HIPBLAS_OP_T, 
                HIPBLAS_OP_N,
                n, n, &alpha,
                d_c, n,
                &beta,
                nullptr, n,
                d_ct, n);

    hipblasGetMatrix(n, n, sizeof(float), d_ct, n, c.data(), n));

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_ct);
    hipblasDestroy(handle);

    return c;
}
