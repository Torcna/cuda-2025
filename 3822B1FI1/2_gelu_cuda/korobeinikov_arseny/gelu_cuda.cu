#include "hip/hip_runtime.h"
#include "gelu_cuda.h"

__global__ void Gelu(float *input, float *output, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        float x = input[i];
        float in_tanh = 0.797885f * (x + 0.044715f * x * x * x);
        output[i] = 0.5f * x * (1.0f + std::tanh(in_tanh));
    }
}

std::vector<float> GeluCUDA(const std::vector<float> &input)
{
    const int size = input.size();
    std::vector<float> output(size);

    float *p_input, *p_output;
    hipMalloc(&p_input, size * sizeof(float));
    hipMalloc(&p_output, size * sizeof(float));

    hipMemcpy(p_input, input.data(), size * sizeof(float), hipMemcpyHostToDevice);

    const int block_size = 256;
    int num_blocks = (input.size() + block_size - 1) / block_size;
    Gelu<<<num_blocks, block_size>>>(p_input, p_output, size);

    hipMemcpy(output.data(), p_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(p_input);
    hipFree(p_output);

    return output;
}